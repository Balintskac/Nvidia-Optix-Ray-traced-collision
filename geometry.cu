#include "hip/hip_runtime.h"
//
// Copyright (c) 2021, NVIDIA CORPORATION. All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
//  * Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
//  * Neither the name of NVIDIA CORPORATION nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
// OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//

#include <optix.h>

#include "optixRayTracedCollision.h"
#include "helpers.h"
#include <iostream>
#include <optixWhitted/helpers.h>

extern "C" {
__constant__ Params params;
}

extern "C" __global__ void __intersection__parallelogram()
{
    const Parallelogram* floor = reinterpret_cast<Parallelogram*>( optixGetSbtDataPointer() );

    const float3 ray_orig = optixGetWorldRayOrigin();
    const float3 ray_dir  = optixGetWorldRayDirection();
    const float ray_tmin = optixGetRayTmin(), ray_tmax = optixGetRayTmax();

    float3 n = make_float3( floor->plane );
    float dt = dot(ray_dir, n );
    float t = (floor->plane.w - dot(n, ray_orig))/dt;
    if( t > ray_tmin && t < ray_tmax )
    {
        float3 p = ray_orig + ray_dir * t;
        float3 vi = p - floor->anchor;
        float a1 = dot(floor->v1, vi);
        if(a1 >= 0 && a1 <= 1)
        {
            float a2 = dot(floor->v2, vi);
            if(a2 >= 0 && a2 <= 1)
            {
                optixReportIntersection(
                    t,
                    0,
                    float3_as_args(n),
                    __float_as_uint( a1 ), __float_as_uint( a2 )
                    );
            }
        }
    }
}

extern "C" __global__ void __intersection__sphere_motion()
{
    const GeometryData::Sphere* sbt_data = reinterpret_cast<GeometryData::Sphere*>(optixGetSbtDataPointer());
    const float3  ray_orig = optixGetWorldRayOrigin();
    const float3  ray_dir = optixGetWorldRayDirection();
    const float ray_tmin = optixGetRayTmin();

    float3 O = ray_orig - sbt_data->center;
    float  l = 1 / length(ray_dir);
    float3 D = ray_dir * l;

    float b = dot(O, D), sqr_b = b * b;
    float O_dot_O = dot(O, O);
    float radius1 = 1.0f;
    float sqr_radius1 = radius1 * radius1;
    float c = O_dot_O - sqr_radius1;
    float root = sqr_b - c;
    if (root > 0.0f) {
        float t1 = -b - sqrtf(root);
        float t2 = -b + sqrtf(root);
        // if (0.0f != length(O)) {
            if (t1 < t2 || t1 == t2) {
                optixReportIntersection(t1 * l, 0);
            }
            else
            {
                optixReportIntersection(t2 * l, 0);
            }
        //  }
        
    }
}

extern "C" __global__ void __intersection__floor_motion()
{
    const Parallelogram* floor = reinterpret_cast<Parallelogram*>(optixGetSbtDataPointer());

    const float3 ray_orig = optixGetWorldRayOrigin();
    const float3 ray_dir = optixGetWorldRayDirection();
    const float ray_tmin = optixGetRayTmin(), ray_tmax = optixGetRayTmax();

    float3 n = make_float3(floor->plane);
    float dt = dot(ray_dir, n);
    float t = (floor->plane.w - dot(n, ray_orig)) / dt;
    if (t > ray_tmin && t < ray_tmax)
    {
        float3 p = ray_orig + ray_dir * t;
        float3 vi = p - floor->anchor;
        float a1 = dot(floor->v1, vi);
        float a2 = dot(floor->v2, vi);

        optixReportIntersection(
            t,
            0,
            float3_as_args(n),
            __float_as_uint(a1), __float_as_uint(a2)
        );
    }
            
        
    
}