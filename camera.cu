#include "hip/hip_runtime.h"
//
// Copyright (c) 2021, NVIDIA CORPORATION. All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
//  * Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
//  * Neither the name of NVIDIA CORPORATION nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
// OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//

#include <hip/hip_vector_types.h>
#include <optix_device.h>
#include "optixRayTracedCollision.h"
#include "random.h"
#include "helpers.h"
#include <cuda/helpers.h>
#include <optixWhitted/helpers.h>

extern "C" {
    __constant__  Params params;
}

extern "C" __global__ void __raygen__pinhole_camera()
{
    const uint3 idx = optixGetLaunchIndex();
    const uint3 dim = optixGetLaunchDimensions();

     CameraData* camera = (CameraData*) optixGetSbtDataPointer();

    const unsigned int image_index = params.width * idx.y + idx.x;
    unsigned int       seed        = tea<16>( image_index, params.subframe_index );

    //589�824 Subpixel jitter: send the ray through a different position inside the pixel each time,
    // to provide antialiasing. The center of each pixel is at fraction (0.5,0.5)
    float2 subpixel_jitter = params.subframe_index == 0 ?
        make_float2(0.5f, 0.5f) : make_float2(rnd(seed), rnd(seed));

    float2 d = ((make_float2(idx.x, idx.y) + subpixel_jitter) / make_float2(params.width, params.height)) * 2.f - 1.f;
    float3 ray_origin = camera->eye;
    float3 ray_direction = normalize(d.x * camera->U + d.y * camera->V + camera->W);

    RadiancePRD prd;
    prd.importance = 1.f;
    prd.depth = 0;

    optixTrace(
        params.handle,
        ray_origin,
        ray_direction,
        params.scene_epsilon,
        1000.f,
        10.0f,
        OptixVisibilityMask(1),
        OPTIX_RAY_FLAG_NONE,
        RAY_TYPE_RADIANCE,
        RAY_TYPE_COUNT,
        RAY_TYPE_RADIANCE,
        float3_as_args(prd.result),
        reinterpret_cast<unsigned int&>(prd.importance),
        reinterpret_cast<unsigned int&>(prd.depth));

    float4 acc_val = params.accum_buffer[image_index];

    if (params.subframe_index > 0)
    {
        acc_val = lerp(acc_val, make_float4(prd.result, 0.f), 1.0f / static_cast<float>(params.subframe_index + 1));
    }
    else
    {
        acc_val = make_float4(prd.result, image_index);
    }

    params.frame_buffer[image_index] = make_color(acc_val);
    params.accum_buffer[image_index] = acc_val;
    
}

extern "C" __global__ void __raygen__sphere_motion()
{
    const uint3 idx = optixGetLaunchIndex();
    const uint3 dim = optixGetLaunchDimensions();

     CameraData* camera = (CameraData*)optixGetSbtDataPointer();
    const unsigned int image_index = params.width_impulse * idx.y + idx.x + idx.z * dim.x * dim.y;

    float PI = 3.14;
    float theta = 2* PI * idx.x/20.f;
    float phi = PI/2 * idx.y/20.f;

    //sin/cos https://www.scratchapixel.com/lessons/mathematics-physics-for-computer-graphics/geometry/spherical-coordinates-and-trigonometric-functions

    float3 ray_direction = {
        cos(phi) * sin(theta),
        cos(theta),
        sin(phi) * sin(theta)
    };

    CollisionPRD prd;
    prd.result = GeometryData::impulseData();
 //   prd.result.impulse = { 0.f };
 //   prd.result.pos = { 0.f };
 //   prd.result.target_pos = { 0.f };
  //  prd.result.target_vel = { 0.f };
    prd.result.ballIndex = idx.z;

    for (int i = 0; i < 32; i++)
       prd.result.ballsInd[i] = 0xffff;

   // float3 ray_origin = camera->spheres.at(idx.z);
    float3 ray_origin = camera->pos[idx.z];
    optixTrace(
        params.handle,
        ray_origin,
        ray_direction,
        params.scene_epsilon,
        100.f,
        2.5f,
        OptixVisibilityMask(1),
        OPTIX_RAY_FLAG_NONE,
        RAY_TYPE_RADIANCE,
        RAY_TYPE_COUNT,
        RAY_TYPE_RADIANCE,
      //  float3_as_args(prd.result.pos),
      //  float3_as_args(prd.result.target_vel),
      //  float3_as_args(prd.result.target_pos),
        reinterpret_cast<unsigned int&>(prd.result.ballIndex),
        reinterpret_cast<unsigned int&>(prd.result.ballsInd)
        );
      
    GeometryData::impulseData acc_val = params.accum_buffer_motion[image_index];

    //   float4 acc = params.accum_buffer[image_index];
        acc_val = prd.result;
      //  acc_val.ballIndex = idx.z * 5000;
      //  acc_val.pos = camera->eye;

    //   acc_val.sourceIndex = image_index;
    //    acc = make_float4(prd.result.impulse, 0.f);
    // Impulse buffer (Ray traed collision)
    params.impulse_buffer[image_index] = acc_val;
    //  params.frame_buffer_motion[image_index] = acc_val.impulse;
    params.accum_buffer_motion[image_index] = acc_val;

    // Render buffer (small window with 100*150)
    //   params.frame_buffer[image_index] = make_color(acc);
    //   params.accum_buffer[image_index] =acc;

}